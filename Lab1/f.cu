
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <cmath>
#include <time.h> 
#include <locale>
#define BLOCK_SIZE  16         

__global__ void matMultCuda(float* a, float* b, float* c, int n ) {
    
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	float sum = 0.0f;
	int ia = n*blockDim.y * blockIdx.y + n*threadIdx.y;
	int ib = blockDim.x * blockIdx.x + threadIdx.x;
	int ic = n * BLOCK_SIZE * blockIdx.y + BLOCK_SIZE * blockIdx.x;
	for ( int k = 0; k < n; k++ ) // calculating the element
		sum += a [ia + k] * b [ib + k*n];
	c [ic + n * ty + tx] = sum; 		
}

//filling with random numbers
void Filling_m(float *A,float *B, int r) {

	srand(time(NULL));
	for(int i=0;i<r; i++)
    		for (int j=0;j<r;j++){
        		A[i * r + j] =  rand()%100 ;
			B[i * r + j] =  rand()%100 ;
		}
}

 

//sequential multiplication of matrices
float matMulCPU(float* a, float* b, float* c, int N)
{
    clock_t start = clock();
    for (int i = 0; i < N; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            c[i * N + j] = 0;
            for (int k = 0; k < N; ++k)
                c[i *N + j] += a[i *N + k] * b[k *N + j];
        }
    }

    clock_t end = clock();
    float ms = ((float)(end - start)/CLOCKS_PER_SEC)*1000;
    return ms;
}


//checking the equality of matrices
int equals_m(float* A, float* B, int n) {
	float e=0.000001;
   	for (int i = 0; i < n * n; i++) {
        	if (A[i] - B[i]>e) {
            	return 0;
        }
    }
    return 1;
}


int main(int argc, char* argv[])
{
	setlocale(LC_ALL, "Russian");
        int N = 512;       // matrix size is N*N
        printf("\n Matrix size: %u \n", N);

	float* a = new float[N * N];
        float* b = new float[N * N];
        float* C_cpu = new float[N * N];
        float* C_gpu = new float[N * N];

	//Fill in with random values
	Filling_m(a,b, N);
		
	//Matrices for CUDA
	float *d_A,*d_B,*d_C;
	hipMalloc ( (void**)&d_A, N * N * sizeof ( float ) );
	hipMalloc ( (void**)&d_B, N * N * sizeof ( float ) );
	hipMalloc ( (void**)&d_C, N * N * sizeof ( float ) );	

	//Event Handler
	hipEvent_t startt, stopp;
    	float gpuTime = 0.0f;
	hipEventCreate(&startt);
    	hipEventCreate(&stopp);

	//Тumber of threads and blocks
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    	dim3 blocks(N/threads.x, N/threads.y);
	
	//Mark the time
	hipEventRecord(startt, 0);

	//Сopy the data to the device
	hipMemcpy(d_A, a, N * N * sizeof(float), hipMemcpyHostToDevice);
    	hipMemcpy(d_B, b, N * N * sizeof(float), hipMemcpyHostToDevice);
	
	//Core function
	matMultCuda<<< blocks, threads >>>(d_A,d_B,d_C,N);
	
	//Get the result
	hipMemcpy(C_gpu, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

	//Stop point
   	hipEventRecord(stopp, 0);
	hipEventSynchronize(stopp);

	hipEventElapsedTime(&gpuTime, startt, stopp);
	double ms = gpuTime;
    	printf("GPU time: %.2f ms\n", ms);

	float ms2=matMulCPU(a, b, C_cpu, N);
      	printf("CPU time: %.2f ms \n",ms2);

	bool yy=equals_m(C_cpu, C_gpu, N);
	printf("Statement: matrices are equal - %s \n", yy ? "true" : "false");
	
        hipHostFree( a);
       	hipHostFree (b);
        hipHostFree (C_cpu);
        hipHostFree (C_gpu);
	hipEventDestroy(startt);
    	hipEventDestroy(stopp);
    	hipFree(d_A);
    	hipFree(d_B);
    	hipFree(d_C);

    return 0;
}

